
#include <hip/hip_runtime.h>
__global__ void sinewave_kernel(float4 *pos, unsigned int width, unsigned int height, float animationTime)
{
	//Code
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	float freq = 4.0f;

	float w = sinf(u * freq + animationTime) * cosf(v * freq + animationTime) * 0.5f;

	pos[y * width + x] = make_float4(u, w, v, 1.0f);
}

void launchCUDAKernel(float4 *pos, unsigned int width, unsigned int height, float animationTime)
{
	//Code
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	sinewave_kernel<<<grid,block>>>(pos, width, height, animationTime);

}