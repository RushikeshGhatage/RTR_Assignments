#include "hip/hip_runtime.h"
//Header
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "helper_timer.h"

//Global Variables
//Odd number 11444777 is deliberate illustration (nVidia OpenCL Samples)
int iNumberOfArrayElements = 5;

float *hostInput1 = NULL;
float *hostInput2 = NULL;
float *hostOutput = NULL;
float *gold = NULL;

float *deviceInput1 = NULL;
float *deviceInput2 = NULL;
float *deviceOutput = NULL;

float timeOnCPU;
float timeOnGPU;

//CUDA KERNEL DEFINITION
// Global Kernel Function Definition
__global__ void vecAdd(float *in1, float *in2, float *out, int len)
{
	//Variable Declaration
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	//Code
	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

int main(int argc, char *argv[])
{
	//Function Declaration
	void fillFloatArrayWithRandomNumbers(float *, int);
	void vecAddHost(const float *, const float *, float *, int);
	void cleanup(void);

	//Code
	//Allocate Host-Memory
	hostInput1 = (float *)malloc(sizeof(float) * iNumberOfArrayElements);
	if (hostInput1 == NULL)
	{
		printf("CPU Memory fatal Error = Cannot Allocate Enough emory For Host Input Array 1.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float *)malloc(sizeof(float) * iNumberOfArrayElements);
	if (hostInput2 == NULL)
	{
		printf("CPU Memory fatal Error = Cannot Allocate Enough emory For Host Input Array 2.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostOutput = (float *)malloc(sizeof(float) * iNumberOfArrayElements);
	if (hostOutput == NULL)
	{
		printf("CPU Memory fatal Error = Cannot Allocate Enough emory For Host Output Array 1.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (float *)malloc(sizeof(float) * iNumberOfArrayElements);
	if (gold == NULL)
	{
		printf("CPU Memory fatal Error = Cannot Allocate Enough emory For Gold Output Array 1.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//Fill Above input host vectors with Random but Hard-Coded data
	fillFloatArrayWithRandomNumbers(hostInput1, iNumberOfArrayElements);
	fillFloatArrayWithRandomNumbers(hostInput2, iNumberOfArrayElements);

	//Allocate Device Memory
	hipError_t err = hipSuccess;
	err = hipMalloc((void **)&deviceInput1, sizeof(float) * iNumberOfArrayElements);
	if (err != hipSuccess)
	{
		printf("GPU Memory fatal Error = %s in file name %s at Line No. %d.\n",hipGetErrorString(err), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&deviceInput2, sizeof(float) * iNumberOfArrayElements);
	if (err != hipSuccess)
	{
		printf("GPU Memory fatal Error = %s in file name %s at Line No. %d.\n",hipGetErrorString(err), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&deviceOutput, sizeof(float) * iNumberOfArrayElements);
	if (err != hipSuccess)
	{
		printf("GPU Memory fatal Error = %s in file name %s at Line No. %d.\n",hipGetErrorString(err), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	//Copy Host Memory Contents to device memory
	err = hipMemcpy(deviceInput1, hostInput1, sizeof(float) * iNumberOfArrayElements, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("GPU Memory fatal Error = %s in file name %s at Line No. %d.\n",hipGetErrorString(err), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(deviceInput2, hostInput2, sizeof(float) * iNumberOfArrayElements, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("GPU Memory fatal Error = %s in file name %s at Line No. %d.\n",hipGetErrorString(err), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	//Cuda Kernel Configuration
	dim3 dimGrid = dim3(ceil(iNumberOfArrayElements / 256.0), 1, 1);
	dim3 dimBlock = dim3(256, 1, 1);

	//Start Timer
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	vecAdd<<<dimGrid,dimBlock>>>(deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements);

	//Stop Timer
	sdkStopTimer(&timer);
	timeOnGPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	// Copy devce memory to host memory
	err = hipMemcpy(hostOutput, deviceOutput, sizeof(float) * iNumberOfArrayElements, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s in File name %s at Line number %d\n",hipGetErrorString(err), __FILE__, __LINE__);
		cleanup();
		exit(EXIT_FAILURE);
	}

	//Results
	vecAddHost(hostInput1, hostInput2, gold, iNumberOfArrayElements);

	//Compare results for Golden-Host
	const float epsilon = 0.000001f;
	bool bAccuracy = true;
	int breakValue = 0;
	int i;
	for(i = 0; i < iNumberOfArrayElements; i++)
	{
		float val1 = gold[i];
		float val2 = hostOutput[i];
		if (fabs(val1 - val2) > epsilon)
		{
			bAccuracy = false;
			breakValue = i;
			break;
		}
	}

	if (bAccuracy == false)
	{
		printf("Break Value = %d\n", breakValue);
	}

	char str[125];
	if (bAccuracy == true)
	{
		sprintf(str, "%s", "Comparison of Output Arrays on CPU and GPU are Accurate within the limit of 0.000001");
	}
	else
	{
		sprintf(str, "%s", "Not All Comparison of Output Arrays on CPU and GPU are Accurate within the limit of 0.000001");	
	}

	printf("1st Array from 0th Element %.6f to %dth element %.6f\n",hostInput1[0], iNumberOfArrayElements - 1 ,hostInput1[iNumberOfArrayElements - 1]);
	printf("2nd Array from 0th Element %.6f to %dth element %.6f\n",hostInput2[0], iNumberOfArrayElements - 1 ,hostInput2[iNumberOfArrayElements - 1]);
	printf("Grid Dimension = (%d, 1, 1) and Block Dimension = (%d, 1, 1)\n", dimGrid.x, dimBlock.x);
	printf("Sum of Each Element from Above 2 Arrays creates 3rd Arrays :\n");
	printf("3rd Array is From 0th Element %.6f to %dth element %.6f\n", hostOutput[0], iNumberOfArrayElements - 1, hostOutput[iNumberOfArrayElements - 1]);
	printf("The Time taken to do Above Addition on CPU = %.6f (ms)\n", timeOnCPU);
	printf("The Time taken to do Above Addition on GPU = %.6f (ms)\n", timeOnGPU);
	printf("%s\n", str);

	//Total Cleanup
	cleanup();
	return(0);
}

void cleanup(void)
{

	//Code

	//Free Allocated Device Memory
	if (deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}


	if (deviceInput2)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if (deviceOutput)
	{
		hipFree(deviceOutput);
		deviceOutput = NULL;
	}

	//Free Allocated Host Memory
	if (hostInput1)
	{
		hipFree(hostInput1);
		hostInput1 = NULL;
	}


	if (hostInput2)
	{
		hipFree(hostInput2);
		hostInput2 = NULL;
	}

	if (hostOutput)
	{
		hipFree(hostOutput);
		hostOutput = NULL;
	}

	if (gold)
	{
		free(gold);
		gold = NULL;
	}
	
}

void fillFloatArrayWithRandomNumbers(float *pFloatArray, int iSize)
{
	//Code
	int i;
	const float fScale = 1.0f / (float)RAND_MAX;
	for (i = 0; i < iSize; ++i)
	{
		pFloatArray[i] = fScale * rand();
	}
}

//"Golden" Host Processing vector addition function for comparison purposes
void vecAddHost(const float* pFloatData1, const float* pFloatData2, float* pFloatResult, int iNumElement)
{
	int i;
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for (i = 0; i < iNumberOfArrayElements; i++)
	{
		pFloatResult[i] = pFloatData1[i] + pFloatData2[i];
	}

	sdkStopTimer(&timer);
	timeOnCPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
}
