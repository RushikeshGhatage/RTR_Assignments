//Header
#include<stdio.h>
#include<hip/hip_runtime.h>

//Global Variable
int inputLength_rvg = 5;

float *hostInput1_rvg = NULL;
float *hostInput2_rvg = NULL;
float *hostOutput_rvg = NULL;

float *deviceInput1_rvg = NULL;
float *deviceInput2_rvg = NULL;
float *deviceOutput_rvg = NULL;

//Global kernel Function definition
__global__ void vecAdd(float *in1, float *in2, float *out, int len)
{
	//Variable Declaration
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	//Code
	if(i <= len)
	{
		out[i] = in1[i] + in2[i];
	}
}

int main(int argc, char *argv[])
{
	//Function Declaration
	void cleanup(void);

	//Code
	//Allocate Host-Memory
	hostInput1_rvg = (float *)malloc(inputLength_rvg * sizeof(float));
	if(hostInput1_rvg == NULL)
	{
		printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Input Array 1.\nExiting ... \n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2_rvg = (float *)malloc(inputLength_rvg * sizeof(float));
	if(hostInput2_rvg == NULL)
	{
		printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Input Array 2.\nExiting ... \n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostOutput_rvg = (float *)malloc(inputLength_rvg * sizeof(float));
	if(hostOutput_rvg == NULL)
	{
		printf("CPU Memory Fatal Error = Can Not Allocate Enough Memory For Host Output Array.\nExiting ... \n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//Fill above input host vectors with arbitary but Hard-Coded Data
	hostInput1_rvg[0] = 101.0;
	hostInput1_rvg[1] = 102.0;
	hostInput1_rvg[2] = 103.0;
	hostInput1_rvg[3] = 104.0;
	hostInput1_rvg[4] = 105.0;

	hostInput2_rvg[0] = 201.0;
	hostInput2_rvg[1] = 202.0;
	hostInput2_rvg[2] = 203.0;
	hostInput2_rvg[3] = 204.0;
	hostInput2_rvg[4] = 205.0;

	//Allocate Device Memory
	int size = inputLength_rvg * sizeof(float);
	hipError_t err = hipSuccess;

	err = hipMalloc((void **)&deviceInput1_rvg,size); 
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s at Line No. %d\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__ );
		cleanup();
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&deviceInput2_rvg,size); 
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s at Line No. %d\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__ );
		hipFree(deviceInput1_rvg);
		cleanup();
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&deviceOutput_rvg,size); 
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s at Line No. %d\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__ );
		cleanup();
		exit(EXIT_FAILURE);
	}


	//Copy Host Memory Contents to Device Memory
	err  = hipMemcpy(deviceInput1_rvg, hostInput1_rvg, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s at Line No. %d\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__ );
		cleanup();
		exit(EXIT_FAILURE);
	}

	err  = hipMemcpy(deviceInput2_rvg, hostInput2_rvg, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s at Line No. %d\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__ );
		cleanup();
		exit(EXIT_FAILURE);
	}

	//Cuda Kernel Configuration
	dim3 DimGrid = dim3(ceil(inputLength_rvg/256.0),1,1);
	dim3 DimBlock = dim3(256, 1,1);
	vecAdd<<<DimGrid, DimBlock>>>(deviceInput1_rvg, deviceInput2_rvg, deviceOutput_rvg, inputLength_rvg);

	//Copy Device Memory to Host Memory
	err  = hipMemcpy(hostOutput_rvg, deviceOutput_rvg, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s at Line No. %d\nExiting ...\n", hipGetErrorString(err), __FILE__, __LINE__ );
		cleanup();
		exit(EXIT_FAILURE);
	}

	//Results
	int i;
	for (i = 0; i < inputLength_rvg; i++)
	{
		printf("%f + %f = %f\n", hostInput1_rvg[i], hostInput2_rvg[i], hostOutput_rvg[i]);
	}

	//Total Cleanup
	cleanup();

	return(0);

}

void cleanup(void)
{

	//Code

	//Free Allocated Device Memory
	if (deviceInput1_rvg)
	{
		hipFree(deviceInput1_rvg);
		deviceInput1_rvg = NULL;
	}


	if (deviceInput2_rvg)
	{
		hipFree(deviceInput2_rvg);
		deviceInput2_rvg = NULL;
	}

	if (deviceOutput_rvg)
	{
		hipFree(deviceOutput_rvg);
		deviceOutput_rvg = NULL;
	}

	//Free Allocated Host Memory
	if (hostInput1_rvg)
	{
		hipFree(hostInput1_rvg);
		hostInput1_rvg = NULL;
	}


	if (hostInput2_rvg)
	{
		hipFree(hostInput2_rvg);
		hostInput2_rvg = NULL;
	}

	if (hostOutput_rvg)
	{
		hipFree(hostOutput_rvg);
		hostOutput_rvg = NULL;
	}
	
}






